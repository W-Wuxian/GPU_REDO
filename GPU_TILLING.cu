#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>


#define SOFT 1e-20f

#define THREADS_PER_BLOCK 128
#define DUMP

typedef struct { 
	float4 *POS, *VIT; 
} PType;


//CUDA VERSION:
__global__ void VitParticles_CUDA(const int nParticles, float4 *p, float4 *v)
{
  // Particle propagation time step
    const float dt = 0.0005f;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i<nParticles){
	float Fx = 0.0, Fy = 0.0, Fz = 0.0;
	for(int stack=0; stack<gridDim.x; stack++){
		__shared__ float3 p3[THREADS_PER_BLOCK];
		float4 tmp_p = p[stack*blockDim.x+threadIdx.x];
		p3[threadIdx.x] = make_float3(tmp_p.x,tmp_p.y,tmp_p.z);
		__syncthreads(); 
		for(int j = 0; j< THREADS_PER_BLOCK; j++){
	  		//avoid singularity and interaction with self:
	  		//const float softening = 1e-20;
	  		//Newton's law of universal gravity:
	  		float dx = p3[j].x - p[i].x;
	  		float dy = p3[j].y - p[i].y;
          		float dz = p3[j].z - p[i].z;
          		float drSquared    = dx*dx + dy*dy + dz*dz + SOFT;
			float invdrS       = rsqrtf(drSquared);
          		float invdrPower32 = invdrS*invdrS*invdrS;
       			  //Calculate the net force:
	  		Fx += dx * invdrPower32;  
          		Fy += dy * invdrPower32;  
          		Fz += dz / invdrPower32;
      		}//j loop
		__syncthreads();
	}	
      	//Accelerate particles in response to the gravitational force:
      	v[i].x += dt*Fx; 
      	v[i].y += dt*Fy; 
      	v[i].z += dt*Fz;
      }
}//fct MoveParticles_CUDA

//-------------------------------------------------------------------------------------------------------------------------

// Initialize random number generator and particles:
void init_rand(int ntol, float *tab){
  srand48(0x2020);
  for (int i = 0; i < ntol; i++)
  {
    tab[i] =  2.0*drand48() - 1.0;
  }
}

//-------------------------------------------------------------------------------------------------------------------------

// Initialize (no random generator) particles
void init_norand(int ntol, const int nParticles , float *tab){
  const float a=127.0/nParticles;
  for (int i = 0; i < ntol; i++)
  {
	if(i>=0&i<nParticles){
    		tab[i] =  i*a;
	}
	if(i>=nParticles&i<2*nParticles){
    		tab[i] =  i*a;
	 }
	 if(i>=2*nParticles&i<3*nParticles){
    		tab[i] =  1.0;
	 }
    	 if(i>=3*nParticles){
    		tab[i] =  0.5;
	 }
  }
}




//-------------------------------------------------------------------------------------------------------------------------

void dump(int iter, int nParticles, float4* p, float4* v)
{
    char filename[64];
        snprintf(filename, 64, "output_%d.txt", iter);

    FILE *f;
        f = fopen(filename, "w+");

    int i;
        for (i = 0; i < nParticles; i++)
	    {
	      fprintf(f, "%e %e %e %e %e %e\n",
		p[i].x, p[i].y, p[i].z, v[i].x, v[i].y, v[i].z);
							      }

    fclose(f);
}
    
//-------------------------------------------------------------------------------------------------------------------------

int main(const int argc, const char** argv)
{
  // Problem size and other parameters
  const int nParticles = (argc > 1 ? atoi(argv[1]) : 16384);
  // Duration of test
  const int nSteps = (argc > 2)?atoi(argv[2]):10;
  // Particle propagation time step
  const float ddt = 0.0005f;
//-------------------------------------------------------------------------------------------------------------------------
  //DEFINE SIZE:
  int SIZE = 2*nParticles * sizeof(float4);
//-------------------------------------------------------------------------------------------------------------------------
  //DECLARATION & ALLOC particle ON HOST:
  float *evo = (float*) malloc(SIZE );
  PType pevo = {(float4*)evo,((float4*)evo)+nParticles};
//-------------------------------------------------------------------------------------------------------------------------
  // Initialize random number generator and particles
  //srand48(0x2020);
  // Initialize random number generator and particles
  //init_rand(nParticles, evo);
  // Initialize (no random generator) particles
  init_norand(8*nParticles, nParticles,evo);
//-------------------------------------------------------------------------------------------------------------------------
  

  // Perform benchmark
  printf("\nPropagating %d particles using 1 thread...\n\n", 
	 nParticles
	 );

  double rate = 0, dRate = 0; // Benchmarking data
  const int skipSteps = 3; // Skip first iteration (warm-up)
  printf("\033[1m%5s %10s %10s %8s\033[0m\n", "Step", "Time, s", "Interact/s", "GFLOP/s"); fflush(stdout);
 int NBR_BLOCKS = (nParticles+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
//-------------------------------------------------------------------------------------------------------------------------
  hipProfilerStart();
//-------------------------------------------------------------------------------------------------------------------------
  float *cuda_evo;
  hipMalloc(&cuda_evo, SIZE);
  PType cuda_pevo = {(float4*)cuda_evo,((float4*)cuda_evo)+nParticles};
//-------------------------------------------------------------------------------------------------------------------------
  for (int step = 1; step <= nSteps; step++) {

    const double tStart = omp_get_wtime(); // Start timing
    hipMemcpy(cuda_evo, evo, SIZE, hipMemcpyHostToDevice);
    VitParticles_CUDA<<<NBR_BLOCKS,THREADS_PER_BLOCK>>>(nParticles, cuda_pevo.POS, cuda_pevo.VIT);
    hipMemcpy(evo, cuda_evo, SIZE, hipMemcpyDeviceToHost);
    const double tEnd = omp_get_wtime(); // End timing

    // Move particles according to their velocities
      // O(N) work, so using a serial loop
        for (int i = 0 ; i < nParticles; i++) {
	    pevo.POS[i].x  += pevo.VIT[i].x*ddt;
	    pevo.POS[i].y  += pevo.VIT[i].y*ddt;
	    pevo.POS[i].z  += pevo.VIT[i].z*ddt;
	}
		     

    const float HztoInts   = ((float)nParticles)*((float)(nParticles-1)) ;
    const float HztoGFLOPs = 20.0*1e-9*((float)(nParticles))*((float)(nParticles-1));

    if (step > skipSteps) { // Collect statistics
      rate  += HztoGFLOPs/(tEnd - tStart); 
      dRate += HztoGFLOPs*HztoGFLOPs/((tEnd - tStart)*(tEnd-tStart)); 
    }

    printf("%5d %10.3e %10.3e %8.1f %s\n", 
	   step, (tEnd-tStart), HztoInts/(tEnd-tStart), HztoGFLOPs/(tEnd-tStart), (step<=skipSteps?"*":""));
    fflush(stdout);

#ifdef DUMP
    dump(step, nParticles, pevo.POS, pevo.VIT);
#endif
  }
  hipFree(cuda_evo);
//-------------------------------------------------------------------------------------------------------------------------
  hipProfilerStop();
//-------------------------------------------------------------------------------------------------------------------------
  rate/=(double)(nSteps-skipSteps); 
  dRate=sqrt(dRate/(double)(nSteps-skipSteps)-rate*rate);
  printf("-----------------------------------------------------\n");
  printf("\033[1m%s %4s \033[42m%10.1f +- %.1f GFLOP/s\033[0m\n",
	 "Average performance:", "", rate, dRate);
  printf("-----------------------------------------------------\n");
  printf("* - warm-up, not included in average\n\n");
  free(evo);
  return 0;
}
